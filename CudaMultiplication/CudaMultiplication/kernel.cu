#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>

#include <stdio.h>

#define N 16



// Kernel definition

__global__ void MatAdd(float A[N][N], float B[N][N],float C[N][N])

{

	int i = threadIdx.x;

	int j = threadIdx.y;

	C[i][j] = A[i][j] + B[i][j];

}

int main()

{

	float *A, *B, *C;

	hipMalloc((void**)&A, sizeof(float) * N * N);

	hipMalloc((void**)&B, sizeof(float) * N * N);

	hipMalloc((void**)&C, sizeof(float) * N * N);

	// Kernel invocation

	dim3 dimBlock(N, N);

	MatAdd << <1, dimBlock >> >((float(*)[16])A, (float(*)[16])B, (float(*)[16])C);

	if (hipGetLastError() != hipSuccess)

		printf("kernel launch failed\n");

	hipDeviceSynchronize();

	if (hipGetLastError() != hipSuccess)

		printf("kernel execution failed\n");

}

/*
Example input for a and b
5 2 6 1
0 6 2 0
3 8 1 4
1 8 5 6
7 5 8 0
1 8 2 6
9 4 3 8
5 3 7 9

Required output of c

96 68 69 69
24 56 18 52
58 95 71 92
90 107 81 142

*/
