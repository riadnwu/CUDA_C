#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include<cstdlib>
#include<conio.h>
#include <stdio.h>
#include <math.h>
#define blockSize 4
using namespace std;

__global__ void MatrixMul(float *ad, float *bd, float *cd, const int n)
{
	unsigned int i = blockSize*blockIdx.x + threadIdx.x;
	unsigned int j = blockSize*blockIdx.y + threadIdx.y;

	for (int k = 0; k<n; k++)
	{
		cd[i + j*n] += ad[j * n + k] * bd[i + k * n];
	}
}

int main()
{
	const int n = 4;
	float ah3[n][n][n], bh3[n][n][n], ch3[n][n][n];
	float ah[n*n][n], bh[n*n][n], ch[n+n][n];
	float *ad, *bd, *cd;

	for (int i = 0; i<n; i++)
	{
		for (int j = 0; j<n; j++)
		{
			for (int k = 0;k < n;k++)
			{
				ah3[i][j][k] = i + 1;
				ah[i+(k+1)*(k + 1)][j] = ah3[i][j][k]; // Covert TMA(3) to G2A
				bh3[i][j][k] = j + 1;
			}
		}
	}

	
// Show G2A
	for (int i = 0; i < n * n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			cout << ah[i][j];
		}

	}

	hipMalloc((void **)&ad, n*n*sizeof(int));
	hipMalloc((void **)&bd, n*n*sizeof(int));
	hipMalloc((void **)&cd, n*n*sizeof(int));


	hipMemcpy(ad, ah, n*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(bd, bh, n*n*sizeof(int), hipMemcpyHostToDevice);


	dim3 dimBlock(blockSize, blockSize, 1);
	dim3 dimGrid(n / blockSize, n / blockSize, 1);

	MatrixMul << <dimGrid, dimBlock >> > (ad, bd, cd, n);

	hipMemcpy(ch, cd, n*n*sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i<n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			cout << ch[i][j] << " ";
		}
		cout << "\n";
	}
	getch();

}


