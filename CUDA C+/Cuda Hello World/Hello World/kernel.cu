#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include<iostream>
#include<conio.h>
using namespace std;

__global__ void Helloword(void)
{
	printf("Hello Word\n");
}

int main()
{
	Helloword <<< 1, 1000 >>>();
	hipDeviceReset();
	getch();
	return 0;
}
