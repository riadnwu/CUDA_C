#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include<cstdlib>
#include<conio.h>
#include <stdio.h>
#include <math.h>
#define blockSize 4
using namespace std;

__global__ void Addition(float *ad, float *bd, float *cd, const int n)
{
	unsigned int i = blockSize*blockIdx.x + threadIdx.x;
	unsigned int j = blockSize*blockIdx.y + threadIdx.y;

	if (i < n && j < n)
	{
		cd[i + j*n] = ad[i + j*n] + bd[i + j*n];
	}
}

int main()
{
	const int n = 4;
	float ah[n][n], bh[n][n], ch[n][n];
	float *ad, *bd, *cd;

	for (int i = 0; i<n; i++)
	{
		for (int j = 0; j<n; j++)
		{
			cin >> ah[i][j];
		}
	}

	for (int i = 0; i<n; i++)
	{
		for (int j = 0; j<n; j++)
		{
			cin >> bh[i][j];
		}
	}


	hipMalloc((void **)&ad, n*n*sizeof(int));
	hipMalloc((void **)&bd, n*n*sizeof(int));
	hipMalloc((void **)&cd, n*n*sizeof(int));


	hipMemcpy(ad, ah, n*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(bd, bh, n*n*sizeof(int), hipMemcpyHostToDevice);


	dim3 dimGrid(n / blockSize, n / blockSize, 1);
	dim3 dimBlock(blockSize, blockSize, 1);

	Addition << <dimGrid, dimBlock >> > (ad, bd, cd, n);

	hipMemcpy(ch, cd, n*n*sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i<n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			cout << ch[i][j] << " ";
		}
		cout << "\n";
	}
	getch();

}

/*
Example input for a and b
5 2 6 1
0 6 2 0
3 8 1 4
1 8 5 6
7 5 8 0
1 8 2 6
9 4 3 8
5 3 7 9

Required output of c

12 7 14 1
1 14 4 6
12 12 4 12
6 11 12 15

*/

