#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include<cstdlib>
#include<conio.h>
#define TILE_WIDTH 2
#include <stdio.h>
#include <math.h>
using namespace std;

/*matrix multiplication kernels*/

//non shared
__global__ void
MatrixMul(float *Md, float *Nd, float *Pd, const int WIDTH)
{

	// calculate thread id

	unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x;

	unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y;

	for (int k = 0; k<WIDTH; k++)
	{
		Pd[row*WIDTH + col] += Md[row * WIDTH + k] * Nd[k * WIDTH + col];
	}
}

// shared
__global__ void
MatrixMulSh(float *Md, float *Nd, float *Pd, const int WIDTH)
{

	//Taking shared array to break the MAtrix in Tile widht and fatch them in that array per ele

	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];

	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

	// calculate thread id
	unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x;
	unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y;

	for (int m = 0; m<WIDTH / TILE_WIDTH; m++) // m indicate number of phase
	{
		Mds[threadIdx.y][threadIdx.x] = Md[row*WIDTH + (m*TILE_WIDTH + threadIdx.x)];
		Nds[threadIdx.y][threadIdx.x] = Nd[(m*TILE_WIDTH + threadIdx.y) * WIDTH + col];

		for (int k = 0; k<TILE_WIDTH; k++)
			Pd[row*WIDTH + col] += Mds[threadIdx.x][k] * Nds[k][threadIdx.y];

	}
}

// main routine
int main()
{
	const int n = 6;
	float array1_h[n][n], array2_h[n][n], result_array_h[n][n], M_result_array_h[n][n];
	float *array1_d, *array2_d, *result_array_d, *M_result_array_d; // device array
	int i, j;
	//input in host array
	for (i = 0; i<n; i++)
	{
		for (j = 0; j<n; j++)
		{
			array1_h[i][j] = 1;
			array2_h[i][j] = 2;
		}
	}

	//create device array hipMalloc ( (void **)&array_name, sizeofmatrixinbytes) ;

	hipMalloc((void **)&array1_d, n*n*sizeof(int));

	hipMalloc((void **)&array2_d, n*n*sizeof(int));



	//copy host array to device array; hipMemcpy ( dest , source , WIDTH , direction )

	hipMemcpy(array1_d, array1_h, n*n*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(array2_d, array2_h, n*n*sizeof(int), hipMemcpyHostToDevice);



	//allocating memory for resultent device array

	hipMalloc((void **)&result_array_d, n*n*sizeof(int));

	hipMalloc((void **)&M_result_array_d, n*n*sizeof(int));



	//calling kernal

	dim3 dimGrid(n / TILE_WIDTH, n / TILE_WIDTH, 1);

	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

	// Change if 0 to if 1 for running non shared code and make if 0 for shared memory code
#if 0

	MatrixMul << <dimGrid, dimBlock >> > (array1_d, array2_d, M_result_array_d, WIDTH);

#endif

#if 1

	MatrixMulSh << <dimGrid, dimBlock >> > (array1_d, array2_d, M_result_array_d, WIDTH);

#endif

	// all gpu function blocked till kernel is working
	//copy back result_array_d to result_array_h

	hipMemcpy(M_result_array_h, M_result_array_d, n*n*sizeof(int),
		hipMemcpyDeviceToHost);

	//printf the result array
	for (i = 0; i<n; i++)
	{
		for (j = 0; j < n; j++)
		{
			printf("%f   ", M_result_array_h[i][j]);
		}
		printf("\n");
	}
	system("pause");
}

/*
Example input for a and b
5 2 6 1
0 6 2 0
3 8 1 4
1 8 5 6
7 5 8 0
1 8 2 6
9 4 3 8
5 3 7 9

Required output of c

96 68 69 69
24 56 18 52
58 95 71 92
90 107 81 142

*/

