#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include<cstdlib>
#include<conio.h>
#include <stdio.h>
#include <math.h>
#define blockSize 4
using namespace std;

__global__ void MatrixMul(float *ad, float *bd, float *cd, const int n)
{
	unsigned int i = blockSize*blockIdx.x + threadIdx.x;
	unsigned int j = blockSize*blockIdx.y + threadIdx.y;

	for (int k = 0; k<n; k++)
	{
		cd[ i+ j*n] += ad[j * n + k] * bd[i + k * n ];
	}
}

int main()
{
	const int n = 4;
	float ah[n][n], bh[n][n], ch[n][n];
	float *ad, *bd, *cd; 

	for (int i = 0; i<n; i++)
	{
		for (int j = 0; j<n; j++)
		{
			cin>>ah[i][j];
		}
	}

	for (int i = 0; i<n; i++)
	{
		for (int j = 0; j<n; j++)
		{
			cin>>bh[i][j];
		}
	}


	hipMalloc((void **)&ad, n*n*sizeof(int));
	hipMalloc((void **)&bd, n*n*sizeof(int));
	hipMalloc((void **)&cd, n*n*sizeof(int));


	hipMemcpy(ad, ah, n*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(bd, bh, n*n*sizeof(int), hipMemcpyHostToDevice);


	dim3 dimBlock(blockSize, blockSize, 1);
	dim3 dimGrid(n / blockSize, n / blockSize, 1);
	
   MatrixMul << <dimGrid, dimBlock >> > (ad, bd, cd, n);

   hipMemcpy(ch, cd, n*n*sizeof(int),hipMemcpyDeviceToHost);

	for (int i = 0; i<n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			cout<< ch[i][j]<<" ";
		}
		cout << "\n";
	}
	getch();

}

/*
Example input for a and b
5 2 6 1
0 6 2 0
3 8 1 4
1 8 5 6
7 5 8 0
1 8 2 6
9 4 3 8
5 3 7 9

Required output of c

96 68 69 69
24 56 18 52
58 95 71 92
90 107 81 142

*/

