#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>
#include<conio.h>
#include<hip/hip_runtime.h>
#include<iostream>
#include<stdlib.h>
using namespace std;



__global__ void AdditionTwoArray(float *da, float*db, float *dc, const int n)
{
	int i = threadIdx.x;
	if (i < n)
	{
		dc[i] = da[i] + db[i];
		printf("Addition :%f*%f=%f\n", da[i], db[i],dc[i]);
	}
		
}



int main()
{
	float *ha, *hb, *hc;
	float *da, *db, *dc;
	const int n = 1024;
	const int ln = n * sizeof(float);

	ha = (float *)malloc(ln);
	hb = (float *)malloc(ln);
	hc = (float *)malloc(ln);

	for (int i = 0;i < n;i++)
	{
		ha[i] = (rand() / (float)RAND_MAX * 19) + 1;
		hb[i] = (rand() / (float)RAND_MAX * 19) + 1;
		hc[i] = 0;
	}

	hipMalloc(&da, ln);
	hipMalloc(&db, ln);
	hipMalloc(&dc, ln);

	hipMemcpy(da, ha, ln, hipMemcpyHostToDevice);
	hipMemcpy(db, hb, ln, hipMemcpyHostToDevice);
	hipMemcpy(dc, hc, ln, hipMemcpyHostToDevice);

	AdditionTwoArray<<<1,n>>> (da, db, dc, n);

	hipMemcpy(hc, dc, ln, hipMemcpyDeviceToHost);

	/*for (int i = 0;i < n;i++)
	{
		//cout << "Addition: " << ha[i] << " + " << hb[i] << " = " << hc[i] << endl;
	}*/

	free(ha);
	free(hb);
	free(hc);
	hipFree(da);
	hipFree(db);
	hipFree(dc);

	hipDeviceReset();
	_getch();
	return 0;
}

